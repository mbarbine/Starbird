#include <hip/hip_runtime.h>


__global__ void apply_gravity(float* position, float* velocity, float gravity, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        velocity[i] += gravity;
        position[i] += velocity[i];
    }
}

extern "C" void run_apply_gravity(float* position, float* velocity, float gravity, int size) {
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    apply_gravity<<<gridSize, blockSize>>>(position, velocity, gravity, size);
    hipDeviceSynchronize();
}
