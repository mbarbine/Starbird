#include <hip/hip_runtime.h>


// CUDA kernel for hyperspace effect
__global__ void hyperspace_effect(unsigned char* screen, int width, int height, int time) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int offset = (y * width + x) * 3;
        screen[offset] = (x + time) % 255;  // Red channel
        screen[offset + 1] = (y + time) % 255;  // Green channel
        screen[offset + 2] = 128;  // Blue channel
    }
}

extern "C" void run_hyperspace_effect(unsigned char* screen, int width, int height, int time) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    hyperspace_effect<<<gridSize, blockSize>>>(screen, width, height, time);
    hipDeviceSynchronize();
}
